#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SEED 10


__device__
inline static float distance(float cluster_x, float cluster_y, float sample_x, float sample_y) {
    return powf(cluster_x - sample_x, 2) + (powf(cluster_y - sample_y, 2));
}

__global__
void calc_closest(
    float *samples_x,
    float *samples_y,
    int n,
    float *clusters_x,
    float *clusters_y,
    int k,
    int *closest_array
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int closest = -1;
    float shortest_dist = __FLT_MAX__;
    for (int i = 0; i < k; i++) {
        float dist = distance(clusters_x[i], clusters_y[i], samples_x[id], samples_y[id]);
        if (dist < shortest_dist) {
            shortest_dist = dist;
            closest = i;
        }
    }
    closest_array[id] = closest;
}

/**
 * @brief Initializes the clusters with the respective first samples
 *
 * @param samples Array of samples
 * @param k Number of clusters to create
 * @return CArray with all the generated clusters
 */
CArray init_clusters(SArray samples, int k) {
    CArray clusters = (CArray) malloc(sizeof(Clusters));

    clusters->x = (float *) malloc(k * sizeof(float));
    clusters->y = (float *) malloc(k * sizeof(float));
    clusters->samples_size = (int *) malloc(k * sizeof(int));

    for (int i = 0; i < k; ++i) { // Complexity: K
        clusters->x[i] = samples->x[i];
        clusters->y[i] = samples->y[i];
        clusters->samples_size[i] = 0;
    }

    return clusters;
}

/**
 * @brief Initializes the samples randomly
 * 
 * @param n Number of samples to create
 * @return SArray with all the generated samples
*/
SArray init_samples(int n) {
    srand(SEED);

    SArray samples = (SArray) malloc(sizeof(Samples));

    samples->x = (float *) malloc(n * sizeof(float));
    samples->y = (float *) malloc(n * sizeof(float));
    samples->cluster = (int *) malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) { // Complexity: N
        samples->x[i] = (float) rand() / RAND_MAX;
        samples->y[i] = (float) rand() / RAND_MAX;
        samples->cluster[i] = -1;
    }

    return samples;
}

/**
 * @brief Assigns the closest centroid to each sample and computes the new centroids
 *
 * @param samples Array of samples
 * @param n Number of samples
 * @param clusters Array of centroids
 * @param k Number of clusters
*/
int compute_samples(SArray samples, int n, CArray clusters, int k) {
    int cluster_changed = 0;
    int sample_sizes[k];
    float clusters_x[k];
    float clusters_y[k];

    for (int i = 0; i < k; ++i) { // Reset samples_size field in all clusters
        sample_sizes[i] = 0;
        clusters_x[i] = 0;
        clusters_y[i] = 0;
    } // Complexity: K

    float *gpu_samples_x, *gpu_samples_y;
    float *gpu_clusters_x, *gpu_clusters_y;
    int *new_closest;
    int samples_bytes = n * sizeof(float);
    int cluster_bytes = k * sizeof(float);
    int out_bytes = n * sizeof(int);

    hipMalloc((void**) &gpu_samples_x, samples_bytes);
    hipMalloc((void**) &gpu_samples_y, samples_bytes);
    hipMalloc((void**) &gpu_clusters_x, cluster_bytes);
    hipMalloc((void**) &gpu_clusters_y, cluster_bytes);
    hipMalloc((void**) &new_closest, out_bytes);

    hipMemcpy(gpu_samples_x, samples->x, samples_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_samples_y, samples->y, samples_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_clusters_x, clusters->x, cluster_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_clusters_y, clusters->y, cluster_bytes, hipMemcpyHostToDevice);
    
    int num_blocks = n / 1000;
    
    calc_closest <<< 1000, num_blocks >>> (gpu_samples_x, gpu_samples_y, n, gpu_clusters_x, gpu_clusters_y, k, new_closest);

    hipMemcpy(samples->cluster, new_closest, out_bytes, hipMemcpyDeviceToHost);

//#pragma omp parallel for reduction(+:sample_sizes, clusters_x, clusters_y) schedule(static)
    for (int i = 0; i < n; i++) { // Complexity: N
/*

        if (samples->cluster[i] != closest) {
            samples->cluster[i] = closest;
            cluster_changed = 1;
        }
*/       
        int closest = samples->cluster[i];
        sample_sizes[closest]++;
        clusters_x[closest] += samples->x[i];
        clusters_y[closest] += samples->y[i];
    }

    for (int i = 0; i < k; i++) {
        clusters->samples_size[i] = sample_sizes[i];

        // Computing the centroids, assigning a new position for each cluster
        clusters->x[i] = clusters_x[i] / sample_sizes[i];
        clusters->y[i] = clusters_y[i] / sample_sizes[i];
    }

    return ++cluster_changed;
}