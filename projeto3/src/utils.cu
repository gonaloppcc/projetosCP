#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include "../include/cp-utils.h"
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

#define SEED 10
#define BLOCK_SIZE 1024

using namespace std;

float *gpu_samples_x, *gpu_samples_y;
float *gpu_clusters_x, *gpu_clusters_y;
int *closest_cluster;
int *changed_gpu;

int samples_bytes; // = n * sizeof(float);
int cluster_bytes; //= k * sizeof(float);
int closest_cluster_bytes; // = n * sizeof(int);
int changed_gpu_bytes; // = BLOCK_SIZE * sizeof(int);


__device__
inline static float distance(float cluster_x, float cluster_y, float sample_x, float sample_y) {
    return powf(cluster_x - sample_x, 2) + (powf(cluster_y - sample_y, 2));
}

__global__
void calc_closest(
        float *samples_x,
        float *samples_y,
        int n,
        float *clusters_x,
        float *clusters_y,
        int k,
        int *closest_array,
        int *changed
) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedArray[BLOCK_SIZE];

    if (gid >= n)
        return;

    int closest = -1;
    float shortest_dist = __FLT_MAX__;
    for (int i = 0; i < k; i++) {
        float dist = distance(clusters_x[i], clusters_y[i], samples_x[gid], samples_y[gid]);
        if (dist < shortest_dist) {
            shortest_dist = dist;
            closest = i;
        }
    }
    sharedArray[threadIdx.x] = closest_array[gid] != closest;
    closest_array[gid] = closest;
    __syncthreads();
    if (threadIdx.x != 0)
        return;

    for (int i = 0; i < blockDim.x; i++) {
        changed[blockIdx.x] = 0;
        changed[blockIdx.x] += sharedArray[i];
    }
}

/**
 * @brief Initializes the clusters with the respective first samples
 *
 * @param samples Array of samples
 * @param k Number of clusters to create
 * @return CArray with all the generated clusters
 */
CArray init_clusters(SArray samples, int k) {
    CArray clusters = (CArray) malloc(sizeof(Clusters));

    clusters->x = (float *) malloc(k * sizeof(float));
    clusters->y = (float *) malloc(k * sizeof(float));
    clusters->samples_size = (int *) malloc(k * sizeof(int));

    cluster_bytes = k * sizeof(float);
    changed_gpu_bytes = BLOCK_SIZE * sizeof(int);

    hipMalloc((void **) &gpu_clusters_x, cluster_bytes);
    hipMalloc((void **) &gpu_clusters_y, cluster_bytes);
    hipMalloc((void **) &changed_gpu, changed_gpu_bytes);

    for (int i = 0; i < k; ++i) { // Complexity: K
        clusters->x[i] = samples->x[i];
        clusters->y[i] = samples->y[i];
        clusters->samples_size[i] = 0;
    }

    return clusters;
}

/**
 * @brief Initializes the samples randomly
 * 
 * @param n Number of samples to create
 * @return SArray with all the generated samples
*/
SArray init_samples(int n) {
    srand(SEED);

    SArray samples = (SArray) malloc(sizeof(Samples));

    samples->x = (float *) malloc(n * sizeof(float));
    samples->y = (float *) malloc(n * sizeof(float));
    samples->cluster = (int *) malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) { // Complexity: N
        samples->x[i] = (float) rand() / RAND_MAX;
        samples->y[i] = (float) rand() / RAND_MAX;
        samples->cluster[i] = -1;
    }

    samples_bytes = n * sizeof(float);
    closest_cluster_bytes = n * sizeof(int);

    hipMalloc((void **) &gpu_samples_x, samples_bytes);
    hipMalloc((void **) &gpu_samples_y, samples_bytes);
    hipMalloc((void **) &closest_cluster, closest_cluster_bytes);

    return samples;
}

/**
 * @brief Assigns the closest centroid to each sample and computes the new centroids
 *
 * @param samples Array of samples
 * @param n Number of samples
 * @param clusters Array of centroids
 * @param k Number of clusters
*/
int compute_samples(SArray samples, int n, CArray clusters, int k) {
    int cluster_changed = 0;
    int sample_sizes[k];
    float clusters_x[k];
    float clusters_y[k];

    for (int i = 0; i < k; ++i) { // Reset samples_size field in all clusters
        sample_sizes[i] = 0;
        clusters_x[i] = 0;
        clusters_y[i] = 0;
    } // Complexity: K

    int changed_cpu[BLOCK_SIZE];

    int num_blocks = n / BLOCK_SIZE;

    startKernelTime();

    hipMemcpy(gpu_samples_x, samples->x, samples_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_samples_y, samples->y, samples_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_clusters_x, clusters->x, cluster_bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_clusters_y, clusters->y, cluster_bytes, hipMemcpyHostToDevice);
    hipMemcpy(closest_cluster, samples->cluster, closest_cluster_bytes, hipMemcpyHostToDevice);


    calc_closest <<< num_blocks, BLOCK_SIZE >>>(gpu_samples_x, gpu_samples_y, n, gpu_clusters_x, gpu_clusters_y, k,
                                                closest_cluster, changed_gpu);

    hipDeviceSynchronize();
    hipMemcpy(samples->cluster, closest_cluster, closest_cluster_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(changed_cpu, changed_gpu, changed_gpu_bytes, hipMemcpyDeviceToHost);

    stopKernelTime();

    for (int i = 0; i < BLOCK_SIZE; i++) {
        cluster_changed += changed_cpu[i];
    }

    for (int i = 0; i < n; i++) { // Complexity: N
        int closest = samples->cluster[i];
        sample_sizes[closest]++;
        clusters_x[closest] += samples->x[i];
        clusters_y[closest] += samples->y[i];
    }

    for (int i = 0; i < k; i++) {
        clusters->samples_size[i] = sample_sizes[i];

        // Computing the centroids, assigning a new position for each cluster
        clusters->x[i] = clusters_x[i] / sample_sizes[i];
        clusters->y[i] = clusters_y[i] / sample_sizes[i];
    }

    return cluster_changed > 0;
}