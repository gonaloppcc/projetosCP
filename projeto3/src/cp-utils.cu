#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "../include/cp-utils.h"

using namespace std;

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (hipEvent_t *start, hipEvent_t *stop) {
	hipEventCreate(start);
	hipEventCreate(stop);

	hipEventRecord(*start);
}

float stopKernelTime (hipEvent_t *start, hipEvent_t *stop) {
	hipEventRecord(*stop);

	hipEventSynchronize(*stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, *start, *stop);

	return milliseconds;
}