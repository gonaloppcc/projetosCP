#include "../include/utils.h"
#include "../include/cp-utils.h"
#include <stdio.h>
#include <stdlib.h>

#define MAX_ITERATIONS 20000

int main(int argc, char *argv[]) {
    if (argc < 3) {
        fprintf(stderr, "Use: kmeans [SAMPLE_NUM] [CLUSTER_NUM] [THREAD_NUM]\n");
        return 1;
    }

    int sample_num = atoi(argv[1]);
    int cluster_num = atoi(argv[2]);

    hipEvent_t start_program, stop_program;
    startKernelTime(&start_program, &stop_program);

    SArray samples = init_samples(sample_num);
    CArray clusters = init_clusters(samples, cluster_num);

    /*
     * Main loop of the program
     * We assign a cluster to each sample and then calculate the new centroid of the cluster
     * Once no sample changes cluster we found the solution and exit the loop
    */
    int changed = 1;
    int iterations = 0;

    float millis_memcpy = 0;
    float millis_kernel = 0;

    while (iterations < MAX_ITERATIONS && changed) {
        changed = compute_samples(samples, sample_num, clusters, cluster_num, &millis_memcpy, &millis_kernel);

        iterations += changed; // If the algorithm has not converged we increment, otherwise iterations stays the same
    }
    float total_execution_time = stopKernelTime(&start_program, &stop_program);

    printf("Total execution time: %.2fms\n\n", total_execution_time);
    printf("Total time spent on memcpy: %.2fms\n", millis_memcpy);
    printf("Average time spent on memcpy: %.2fms\n\n", millis_memcpy / iterations);
    printf("Total time spent on kernel execution: %.2fms\n", millis_kernel);
    printf("Average time spent on kernel execution: %.2fms\n\n", millis_kernel / iterations);
    printf("N = %d, K = %d\n", sample_num, cluster_num);
    for (int i = 0; i < cluster_num; ++i) {
        printf("Center: (%.3f, %.3f) : Size: %d\n", clusters->x[i], clusters->y[i], clusters->samples_size[i]);
    }
    printf("Iterations: %d\n", iterations);

    return 0;
}
